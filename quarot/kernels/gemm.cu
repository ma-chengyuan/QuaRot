#include "hip/hip_runtime.h"
#include <cstdint>
#include <cutlass/gemm/device/gemm.h>
#include <gemm.h>

namespace {

struct MatMulParams {
    constexpr static size_t N_THR = 256;

    constexpr static size_t SHM_M = 128;
    constexpr static size_t SHM_N = 256;
    constexpr static size_t SHM_K = 128;

    constexpr static size_t CPY_K = 32; // Copy at the granularity of 16 bytes

    constexpr static size_t MMA_M = 16;
    constexpr static size_t MMA_N = 8;
    constexpr static size_t MMA_K = 64;

    constexpr static size_t REG_M = 4;
    constexpr static size_t REG_N = 8;
};

template <typename A, typename B> auto ceil_div(A a, B b) { return (a + b - 1) / b; }

template <typename P>
constexpr size_t SHM_K_STRIDE = [] {
    constexpr size_t EPB = 2; // Elements per byte
    static_assert(P::SHM_K % (EPB * sizeof(int32_t)) == 0);
    constexpr size_t SHM_K_INT32S = P::SHM_K / (EPB * sizeof(int32_t));
    static_assert(SHM_K_INT32S >= 4);
    return (((SHM_K_INT32S - 4 + 7) & ~7) + 4) * sizeof(int32_t);
}();

template <typename P> constexpr size_t SHM_A_SIZE = P::SHM_M * SHM_K_STRIDE<P>;
template <typename P> constexpr size_t SHM_B_SIZE = P::SHM_N * SHM_K_STRIDE<P>;
template <typename P> constexpr size_t SHM_SIZE = 2 * (SHM_A_SIZE<P> + SHM_B_SIZE<P>);

template <typename T> __device__ void async_copy(T *dst, const T *src) {
    static_assert(sizeof(T) == 16 || sizeof(T) == 8 || sizeof(T) == 4);
    const uint32_t dst_smem = static_cast<uint32_t>(__cvta_generic_to_shared(dst));
    if constexpr (sizeof(T) == 16) {
        asm volatile("cp.async.cg.shared.global.L2::128B [%0], [%1], 16;" ::"r"(dst_smem), "l"(src));
    } else if constexpr (sizeof(T) == 8) {
        asm volatile("cp.async.ca.shared.global [%0], [%1], 8;" ::"r"(dst_smem), "l"(src));
    } else if constexpr (sizeof(T) == 4) {
        asm volatile("cp.async.ca.shared.global [%0], [%1], 4;" ::"r"(dst_smem), "l"(src));
    }
}

__device__ void async_copy_waitall() { asm volatile("cp.async.wait_all;\n" ::); }

template <typename T> __device__ void swap(T &a, T &b) {
    T tmp = a;
    a = b;
    b = tmp;
}

template <typename P>
__global__ __launch_bounds__(P::N_THR) void matmul_handwritten(const uint8_t *A, const uint8_t *B, int32_t *C,
                                                               uint32_t M, uint32_t N, uint32_t K) {
    constexpr size_t EPB = 2; // Elements per byte

    static_assert(P::N_THR % 32 == 0, "N_THR must be a multiple of 32");

    static_assert(P::SHM_K % P::CPY_K == 0, "SHM_K must be a multiple of CPY_K");
    // Either partial copy or full tiled copy
    static_assert(P::N_THR * P::CPY_K >= P::SHM_M * P::SHM_K ||
                      P::SHM_M * P::SHM_K % (P::N_THR * P::CPY_K) == 0,
                  "N_THR * CPY_K must be >= SHM_M * SHM_K");
    static_assert(P::N_THR * P::CPY_K >= P::SHM_N * P::SHM_K ||
                      P::SHM_N * P::SHM_K % (P::N_THR * P::CPY_K) == 0,
                  "N_THR * CPY_K must be >= SHM_N * SHM_K");

    // clang-format off
    using CopyType = std::conditional_t<P::CPY_K ==  4 * EPB, uint32_t,
                     std::conditional_t<P::CPY_K ==  8 * EPB, uint64_t,
                     std::conditional_t<P::CPY_K == 16 * EPB, float4, void>>>;
    // clang-format on

    constexpr size_t N_WRP = P::N_THR / 32;
    static_assert(P::SHM_M % (P::REG_M * P::MMA_M) == 0 && P::SHM_N % (P::REG_N * P::MMA_N) == 0, "");
    // Number of warp tiles along M and N
    constexpr size_t WRP_M = P::SHM_M / (P::REG_M * P::MMA_M);
    constexpr size_t WRP_N = P::SHM_N / (P::REG_N * P::MMA_N);
    static_assert(WRP_M * WRP_N == N_WRP, "");

    extern __shared__ uint8_t smem[];
    uint8_t *__restrict__ smem_A = smem;
    uint8_t *__restrict__ smem_B = smem + SHM_A_SIZE<P> * 2;

    const size_t tid = threadIdx.x;
    const size_t wid = tid / 32;
    const size_t lid = tid % 32;
    const size_t gid = lid / 8;

    uint8_t *__restrict__ smem_A_cur = smem_A, *__restrict__ smem_A_next = smem_A + SHM_A_SIZE<P>;
    uint8_t *__restrict__ smem_B_cur = smem_B, *__restrict__ smem_B_next = smem_B + SHM_B_SIZE<P>;

#define unroll _Pragma("unroll")
    const auto async_copy_to_smem = [&](size_t k) {
        unroll for (size_t i = 0; i < P::SHM_M * P::SHM_K; i += P::N_THR * P::CPY_K) {
            // Copy to smem_A
            const size_t tile_idx = i + tid * P::CPY_K;
            if constexpr (P::N_THR * P::CPY_K > P::SHM_M * P::SHM_K) {
                if (tile_idx >= P::SHM_M * P::SHM_K) {
                    break;
                }
            }
            const size_t tile_m = tile_idx / P::SHM_K;
            const size_t gmem_m = tile_m + blockIdx.x * P::SHM_M;
            const size_t tile_k = tile_idx % P::SHM_K;
            const size_t gmem_k = k + tile_k;

            const CopyType *src = reinterpret_cast<const CopyType *>(A + gmem_m * (K / EPB) + gmem_k / EPB);
            CopyType *dst =
                reinterpret_cast<CopyType *>(smem_A_next + tile_m * SHM_K_STRIDE<P> + tile_k / EPB);
            async_copy(dst, src);
        }
        unroll for (size_t i = 0; i < P::SHM_N * P::SHM_K; i += P::N_THR * P::CPY_K) {
            // Copy to smem_B
            const size_t tile_idx = i + tid * P::CPY_K;
            if constexpr (P::N_THR * P::CPY_K > P::SHM_N * P::SHM_K) {
                if (tile_idx >= P::SHM_N * P::SHM_K) {
                    break;
                }
            }
            const size_t tile_n = tile_idx / P::SHM_K;
            const size_t gmem_n = tile_n + blockIdx.y * P::SHM_N;
            const size_t tile_k = tile_idx % P::SHM_K;
            const size_t gmem_k = k + tile_k;
            const CopyType *src = reinterpret_cast<const CopyType *>(B + gmem_n * (K / EPB) + gmem_k / EPB);
            CopyType *dst =
                reinterpret_cast<CopyType *>(smem_B_next + tile_n * SHM_K_STRIDE<P> + tile_k / EPB);
            async_copy(dst, src);
        }
    };

    int32_t c[P::REG_M][P::REG_N][4] = {0};
    const size_t mma_wrp_m = wid / WRP_N * P::REG_M * P::MMA_M;
    const size_t mma_wrp_n = wid % WRP_N * P::REG_N * P::MMA_N;
    const size_t mma_trd_m_ld_base = mma_wrp_m + lid % 8 + (gid % 2) * 8;
    const size_t mma_trd_n_ld_base = mma_wrp_n + lid % 8;

    async_copy_to_smem(0);

    for (size_t k = 0; k < K; k += P::SHM_K) {
        swap(smem_A_cur, smem_A_next);
        swap(smem_B_cur, smem_B_next);
        async_copy_waitall();
        __syncthreads();
        if (k + P::SHM_K < K) {
            async_copy_to_smem(k + P::SHM_K);
        }
        unroll for (size_t k_ = 0; k_ < P::SHM_K; k_ += P::MMA_K) {
            int32_t a[P::REG_M][4], b[P::REG_N][2];
            unroll for (size_t i = 0; i < P::REG_M; i++) {
                const size_t mma_trd_m = mma_trd_m_ld_base + i * P::MMA_M;
                const size_t mma_trd_k = k_ + (gid / 2) * 32;
                const size_t addr =
                    __cvta_generic_to_shared(smem_A_cur + mma_trd_m * SHM_K_STRIDE<P> + mma_trd_k / EPB);
                asm("ldmatrix.sync.aligned.m8n8.x4.shared.b16 {%0, %1, %2, %3}, [%4];"
                    : "=r"(a[i][0]), "=r"(a[i][1]), "=r"(a[i][2]), "=r"(a[i][3])
                    : "l"(addr));
            }
            unroll for (size_t j = 0; j < P::REG_N; j++) {
                const size_t mma_trd_n = mma_trd_n_ld_base + j * P::MMA_N;
                const size_t mma_trd_k = k_ + gid * 32;
                const size_t addr =
                    __cvta_generic_to_shared(smem_B_cur + mma_trd_n * SHM_K_STRIDE<P> + mma_trd_k / EPB);
                asm("ldmatrix.sync.aligned.m8n8.x2.shared.b16 {%0, %1}, [%2];"
                    : "=r"(b[j][0]), "=r"(b[j][1])
                    : "l"(addr));
            }
            unroll for (size_t i = 0; i < P::REG_M; i++) {
                unroll for (size_t j = 0; j < P::REG_N; j++) {
                    asm("mma.sync.aligned.m16n8k64.row.col.s32.s4.s4.s32"
                        " {%0, %1, %2, %3},"
                        " {%4, %5, %6, %7},"
                        " {%8, %9},"
                        " {%10, %11, %12, %13};"
                        : "=r"(c[i][j][0]), "=r"(c[i][j][1]), "=r"(c[i][j][2]), "=r"(c[i][j][3])
                        : "r"(a[i][0]), "r"(a[i][1]), "r"(a[i][2]), "r"(a[i][3]), "r"(b[j][0]), "r"(b[j][1]),
                          "r"(c[i][j][0]), "r"(c[i][j][1]), "r"(c[i][j][2]), "r"(c[i][j][3]));
                }
            }
        }
    }
    unroll for (size_t i = 0; i < P::REG_M; i++) {
        unroll for (size_t j = 0; j < P::REG_N; j++) {
            const size_t mma_trd_m = mma_wrp_m + i * P::MMA_M + lid / 4;
            const size_t mma_trd_n = mma_wrp_n + j * P::MMA_N + (lid % 4) * 2;
            const size_t gmem_m = mma_trd_m + blockIdx.x * P::SHM_M;
            const size_t gmem_n = mma_trd_n + blockIdx.y * P::SHM_N;
            assert(gmem_m < M && gmem_n < N);
            C[(gmem_m + 0) * N + gmem_n + 0] = c[i][j][0];
            C[(gmem_m + 0) * N + gmem_n + 1] = c[i][j][1];
            C[(gmem_m + 8) * N + gmem_n + 0] = c[i][j][2];
            C[(gmem_m + 8) * N + gmem_n + 1] = c[i][j][3];
        }
    }
#undef unroll
}

} // namespace

void matmul_host_fused(const Int4Storage *A, const Int4Storage *B, uint32_t M, uint32_t N, uint32_t K,
                       int32_t *C) {
    using P = MatMulParams;

    const dim3 dim_block{P::N_THR};
    const dim3 dim_grid(ceil_div(M, P::SHM_M), ceil_div(N, P::SHM_N));
    constexpr size_t shmem_size = SHM_SIZE<P>;
    if constexpr (shmem_size > 48 * 1024) {
        ensure(hipFuncSetAttribute(reinterpret_cast<const void*>(matmul_handwritten<P>), hipFuncAttributeMaxDynamicSharedMemorySize,
                                    shmem_size) == hipSuccess,
               "Failed to set shared memory size");
    }
    matmul_handwritten<P><<<dim_grid, dim_block, shmem_size>>>(
        reinterpret_cast<const uint8_t *>(A), reinterpret_cast<const uint8_t *>(B), C, M, N, K);
    ensure(hipDeviceSynchronize() == hipSuccess, "Failed to synchronize device");
}

void matmul_host(const Int4Storage *A, const Int4Storage *B, uint32_t M, uint32_t N, uint32_t K, int32_t *C) {
    using Gemm =
        cutlass::gemm::device::Gemm<cutlass::int4b_t,               // ElementA
                                    cutlass::layout::RowMajor,      // LayoutA
                                    cutlass::int4b_t,               // ElementB
                                    cutlass::layout::ColumnMajor,   // LayoutB
                                    int32_t,                        // ElementOutput
                                    cutlass::layout::RowMajor,      // LayoutOutput
                                    int32_t,                        // ElementAccumulator
                                    cutlass::arch::OpClassTensorOp, // tag indicating Tensor Cores
                                    cutlass::arch::Sm80 // tag indicating target GPU compute architecture //
                                                        // TODO: This is just for compiling on my laptop
                                                        // temporarily. Should be higher when doing
                                                        // benchmarking.
                                    >;

    Gemm gemmOp;

    using GemmCoord = cutlass::gemm::GemmCoord;

    typename Gemm::Arguments arguments{{static_cast<GemmCoord::Index>(M), static_cast<GemmCoord::Index>(N),
                                        static_cast<GemmCoord::Index>(K)},
                                       {(cutlass::int4b_t *)A, K},
                                       {(cutlass::int4b_t *)B, K},
                                       {C, N},
                                       {C, N},
                                       {1, 0}};

    auto status = gemmOp(arguments);

    ensure(status == cutlass::Status::kSuccess, cutlassGetStatusString(status));
}